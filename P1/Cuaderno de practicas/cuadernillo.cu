#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <ostream>

// macro de manejo de errores
#include <stdio.h>
#include <assert.h>
inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

//ejemplo de suma de vectores
//#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h> //CUDA

//------------------

// void CPUFunction()
// {
//   printf("This function is defined to run on the CPU.\n");
// }

// __global__ void GPUFunction()
// {
//   printf("This function is defined to run on the GPU.\n");
// }

//SUMA DE VECTORES

double wtime(void)
{
        static struct timeval   tv0;
        double time_;

        gettimeofday(&tv0,(struct timezone*)0);
        time_=(double)((tv0.tv_usec + (tv0.tv_sec)*1000000));
        return( time_/1000000);
}


void vecAdd(float* A, float* B, float* C,
   int n)
{
	int i;
	for (i = 0; i < n; i++)
		C[i] = A[i] + B[i];
}


__global__ 
void vecAdd_GPU(float* A, float* B, float* C,
   int n)
{
	int i;
	i = threadIdx.x + blockDim.x * blockIdx.x;
	if(i<n) 
		C[i] = A[i] + B[i];
}

int main(int argc, char *argv[])
{

//   CPUFunction();
//   GPUFunction<<<1, 1>>>();
//   /*
//  * The macro can be wrapped around any function returning
//  * a value of type `hipError_t`.
//  */
//   checkCuda( hipDeviceSynchronize() );
//--------------------------------------------------------------------------
//EJEMPLO DE SUMA DE VECTORES

	float *a, *b, *c, *c_host;
	float *a_GPU, *b_GPU, *c_GPU;

	int i, N;

	double t0, t1;


	if(argc>1) {
		N = atoi(argv[1]); printf("N=%i\n", N);
	} else {
		printf("Error!!!! \n ./exec number\n");
	return (0);
	}

	// Mallocs CPU
	a  = (float *)malloc(sizeof(float)*N);
	b  = (float *)malloc(sizeof(float)*N);
	c  = (float *)malloc(sizeof(float)*N);
	c_host  = (float *)malloc(sizeof(float)*N);
	for (i=0; i<N; i++){ a[i] = i-1; b[i] = i;}

	/*****************/
	/* Add Matrix CPU*/
	/*****************/
	t0 = wtime();
	vecAdd(a, b, c, N);
	t1 = wtime(); printf("Time CPU=%f\n", t1-t0);

	// Get device memory for A, B, C
	// copy A and B to device memory
	hipMalloc((void **) &a_GPU, N*sizeof(float));
	hipMemcpy(a_GPU, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **) &b_GPU, N*sizeof(float));
	hipMemcpy(b_GPU, b, N*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void **) &c_GPU, N*sizeof(float));

	// Kernel execution in device
	// (vector add in device)
	dim3 DimBlock(256); // 256 thread per block
	dim3 DimGrid(ceil(N/256.0)+1);
	t0 = wtime();
	vecAdd_GPU<<<DimGrid,DimBlock>>>(a_GPU, b_GPU, c_GPU, N);
	checkCuda(hipDeviceSynchronize());
	t1 = wtime(); printf("Time GPU=%f\n", t1-t0);

	// copy C to host memory
	hipMemcpy(c_host, c_GPU, N*sizeof(float), hipMemcpyDeviceToHost);

	/************/
	/* Results  */
	/************/
	for (i=0; i<N; i++)
		if(fabs(c[i]-c_host[i])>1e-5){
			printf("c!=c_host in (%i): ", i);
			printf("C[%i] = %f C_GPU[%i]=%f\n", i, c[i], i, c_host[i] );
		}

	/* Free CPU */
	free(a);
	free(b);
	free(c);
	free(c_host);

	hipFree(a_GPU); hipFree(b_GPU); hipFree(c_GPU);

	return(1);
}
